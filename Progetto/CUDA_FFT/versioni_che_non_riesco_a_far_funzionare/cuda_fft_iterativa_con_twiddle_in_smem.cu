#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define DR_WAV_IMPLEMENTATION
#include "dr_wav.h"

#define SAMPLE_RATE 44100
//#define PI 3.14159265358979323846
#define PI 3.14159265359f


#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

typedef struct {
    float real;
    float imag;
} complex;

__host__ __device__ complex prodotto_tra_complessi(complex a, complex b) {
    complex result;

    result.real = a.real*b.real - a.imag*b.imag;
    result.imag = a.real*b.imag + a.imag*b.real;

    return result;
}

double cpuSecond() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return ((double)ts.tv_sec + (double)ts.tv_nsec * 1.e-9);
}

void checkResult(complex *hostRef, complex *gpuRef, const int N) {
    // epsilon molto largo. Non so perchè la versione GPU differisce rispetto a quella CPU verso la quinta cifra decimale
    double epsilon = 1.0E-4;    
    bool match = 1;

    for (int i = 0; i < N; i++) {
        if (fabs(hostRef[i].real - gpuRef[i].real) > epsilon || fabs(hostRef[i].imag - gpuRef[i].imag) > epsilon) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host (%f; %f) gpu (%f; %f) at current %d\n", hostRef[i].real, hostRef[i].imag, gpuRef[i].real, gpuRef[i].imag, i);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
}




// Funzione strana che ho trovato. Mi permette di ottenere il bit reverse order degli indici
// dei campioni della trasformata in maniera efficiente O(log n), rispetto all'usare un ciclo O(n)
//
// es. indice a 8 bit = 5:
//      5 = 00000101   ->  reversed = 10100000 = 160 
__host__ __device__ uint32_t reverse_bits(uint32_t x) {
    // 1. Swap the position of consecutive bits
    // 2. Swap the position of consecutive pairs of bits
    // 3. Swap the position of consecutive quads of bits
    // 4. Continue this until swapping the two consecutive 16-bit parts of x
    
    /*
        Primo scambio:
        0xaaa... = 1010-1010-... = bit pari; 0x555... = 0101-0101... bit dispari; 
        - nel primo gruppo seleziono i bit pari e li sposto a destra di una posizione
        - nel secondo gruppo selezioni i bit dispari e li sposto a sinistra di una posizione
        - facendo infine l'or dei due gruppi ottengo i la stringa di bit con le posizioni pari e dispari scambiate

        Lo stesso procedimento si ripete sotto
    */
    x = ((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1);  
    // 0xccc... = 1100-1100-... = coppie di bit pari; 0x333... = 0011-0011... coppie di bit dispari; 
    x = ((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2);
    // 0xf0f0... = 11110000-... = quadruple di bit pari; 0x0f0f... = 00001111 ... quadruple di bit dispari; 
    x = ((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4);
    // stessa cosa con gruppi da 8
    x = ((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8);
    return (x >> 16) | (x << 16);
}

void convert_to_complex(short *input, complex *output, int N) {
    for (int i = 0; i < N; i++) {
        output[i].real = (float)input[i];
        output[i].imag = 0.0;
    }
}

void convert_to_short(complex *input, short *output, int N) {
    for (int i = 0; i < N; i++) {
        output[i] = (short)round(input[i].real); 
    }
}



int fft_iterativa(complex *input, complex *output, int N) {
    // N & (N - 1) = ...01000... & ...00111... = 0
    if (N & (N - 1)) {
        fprintf(stderr, "N=%u deve essere una potenza di due\n", N);

        return -1;
    }

    // num_stadi = "quante volte posso dividere N per due"
    int num_stadi = (int) log2f((float) N);


    double start = cpuSecond();
    for (uint32_t i = 0; i < N; i++) {
        uint32_t rev = reverse_bits(i);
        rev = rev >> (32 - num_stadi);

        if(input == output) {
            if (i < rev) {  
                complex temp = input[i];
                output[i] = input[rev];
                output[rev] = temp;
            }
        }
        else {
            output[i] = input[rev];
        }
    }

    // Stadi 1, ..., log_2(N)
    for (int stadio = 1; stadio <= num_stadi; stadio++) {
        int N_stadio_corrente = 1 << stadio;
        int N_stadio_corrente_mezzi = N_stadio_corrente / 2;

        // Itera sull'array di output con passi pari a N_stadio_corrente
        // k = indice (denormalizzato) del blocco di farfalle considerato nell'array di output 
        for (uint32_t k = 0; k < N; k += N_stadio_corrente) {
            // Calcolo due campioni alla volta per cui itero fino a N_stadio_corrente_mezzi
            for (int j = 0; j < N_stadio_corrente_mezzi; j++) {
                float phi = (-2*PI/N_stadio_corrente) * j; 
                complex twiddle_factor = {
                    cos(phi),
                    sin(phi)
                };

                printf("\tCPU - farfalla %d - stadio %d\n\t\ttwiddle: (%f, %f)\n", k+j, stadio, twiddle_factor.real, twiddle_factor.imag);

                complex a = output[k + j];
                complex b = prodotto_tra_complessi(twiddle_factor, output[k + j + N_stadio_corrente_mezzi]);

                // calcolo trasformata
                output[k + j].real = a.real + b.real;
                output[k + j].imag = a.imag + b.imag;
                // simmetria per la seconda metà
                output[k + j + N_stadio_corrente_mezzi].real = a.real - b.real;
                output[k + j + N_stadio_corrente_mezzi].imag = a.imag - b.imag;
            }
        }
    }

    return EXIT_SUCCESS;
}

int ifft_iterativa(complex *input, complex *output, int N) {
    if (N & (N - 1)) {
        fprintf(stderr, "N=%u deve essere una potenza di due\n", N);

        return -1;
    }

    int num_stadi = (int) log2f((float) N);

    // stadio 0
    for (uint32_t i = 0; i < N; i++) {
        uint32_t rev = reverse_bits(i);
        rev = rev >> (32 - num_stadi);

        /*
            Per comodità ho aggiunto questo controllo che mi permette di fare delle
            trasformazioni inplace
        */
        if(input == output) {
            if (i < rev) {  
                complex temp = input[i];
                output[i] = input[rev];
                output[rev] = temp;
            }
        }
        else {
            output[i] = input[rev];
        }
    }

    // Stadi 1, ..., log_2(N)
    for (int stadio = 1; stadio <= num_stadi; stadio++) {
        int N_stadio_corrente = 1 << stadio;
        int N_stadio_corrente_mezzi = N_stadio_corrente / 2;

        for (uint32_t k = 0; k < N; k += N_stadio_corrente) {
            for (int j = 0; j < N_stadio_corrente_mezzi; j++) {
                float phi = 2*PI/N_stadio_corrente * j;   // segno + per ifft 
                complex twiddle_factor = {
                    cos(phi),
                    sin(phi)
                };

                complex a = output[k + j];
                complex b = prodotto_tra_complessi(twiddle_factor, output[k + j + N_stadio_corrente_mezzi]);

                // calcolo antitrasformata
                output[k + j].real = a.real + b.real;
                output[k + j].imag = a.imag + b.imag;
                // simmetria per la seconda metà
                output[k + j + N_stadio_corrente_mezzi].real = a.real - b.real;
                output[k + j + N_stadio_corrente_mezzi].imag = a.imag - b.imag;
            }
        }
    }

    // normalizza i risultati alla fine
    for(int i=0; i<N; i++) {
        output[i].real /= N;
        output[i].imag /= N;
    }

    return EXIT_SUCCESS;
}





__global__ void fft_bit_reversal(complex *input, complex *output, int N, int num_stadi) {
    uint32_t thread_id = blockIdx.x*blockDim.x + threadIdx.x;

    // controllo se ci sono dei thread in eccesso
    if (thread_id >= N) {
        // printf("\tsono un thread in eccesso\n");
        return;
    }

    // Copia input nell'output con bit-reversal (stadio 0)
    uint32_t rev = reverse_bits(thread_id);
    rev = rev >> (32 - num_stadi);

    if(input == output) {
        if (thread_id < rev) {  
            complex temp = input[thread_id];
            output[thread_id] = input[rev];
            output[rev] = temp;
        }
    }
    else {
        output[thread_id] = input[rev];
    }
}

// Kernel che calcola una farfalla e la sua simmetrica 
__global__ void fft_stage(complex *output, int N, int N_stadio_corrente, int N_stadio_corrente_mezzi,
                          int num_stadi, int stadio_corrente, complex* d_twiddle_factor_array) {

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;

    // controllo se ci sono dei thread in eccesso
    if (thread_id >= N/2) {
        return;
    }

    extern __shared__ complex local_twiddle_factor_array[]; // grande N/2 (le altre N/2 rotazioni sono simmetriche)

    int k = (thread_id / N_stadio_corrente_mezzi) * N_stadio_corrente;
    int j = thread_id % N_stadio_corrente_mezzi;

    // carico la smem con il twiddle che questo thread dovrà usare
    int twiddle_index = j * (1 << (num_stadi-stadio_corrente));
    local_twiddle_factor_array[threadIdx.x] = d_twiddle_factor_array[twiddle_index];
    __syncthreads();

    printf("\tGPU - farfalla %d - stadio %d\n\t\ttwiddle: (%f, %f)\n",
           thread_id, stadio_corrente, d_twiddle_factor_array[twiddle_index].real, d_twiddle_factor_array[twiddle_index].imag);

    // printf("\tGPU - farfalla %d - stadio %d\n\t\ttwiddle: (%f, %f)\n",
    //        thread_id, stadio_corrente, local_twiddle_factor_array[threadIdx.x].real, local_twiddle_factor_array[threadIdx.x].imag);
    
    /*
        float phi = (-2.0f*PI/N_stadio_corrente) * j;

        Ad ogni stadio l'angolo del twiddle raddoppia di dimensione:
            -> "twiddle_factor_array" va quindi indicizzato con j * 2^(num_stadi - stadio_corrente)
    */

    complex a = output[k + j];
    complex b = prodotto_tra_complessi(local_twiddle_factor_array[threadIdx.x], output[k + j + N_stadio_corrente_mezzi]);

    output[k + j].real = a.real + b.real;
    output[k + j].imag = a.imag + b.imag;
    // simmetria
    output[k + j + N_stadio_corrente_mezzi].real = a.real - b.real;
    output[k + j + N_stadio_corrente_mezzi].imag = a.imag - b.imag;
}

void precalcola_twiddle_factors(int N, complex* twiddle_factor_array) {
    // l'incremento minimo è quello dell'ultimo stadio
    float phi_increment = -2.0f*PI/N; 

    for(int i=0; i < N/2; i++) {
        twiddle_factor_array[i].real = (float)cos(phi_increment * i);
        twiddle_factor_array[i].imag = (float)sin(phi_increment * i);
    }
}

double fft_iterativa_cuda(complex *input, complex *output, int N) {
    // Controllo che N sia una potenza di 2
    if (N & (N - 1)) {
        fprintf(stderr, "N=%u deve essere una potenza di due\n", N);
        return 0;
    }

    int num_stadi = (int)log2f((double)N);

    // Alloca memoria sulla GPU
    complex *d_input;
    complex *d_output;
    hipMalloc(&d_output, N*sizeof(complex));
    hipMalloc(&d_input, N*sizeof(complex));
    hipMemcpy(d_input, input, N*sizeof(complex), hipMemcpyHostToDevice);

    // Configurazione dei blocchi e dei thread per il bit reversal
    int threads_per_block = 256;
    int num_threads = N;
    int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;

    double start = cpuSecond();
    // stadio 0
    fft_bit_reversal<<<num_blocks, threads_per_block>>>(d_input, d_output, N, num_stadi);

    // precalcolo dei twiddle factor
    int twiddle_factor_array_size = (N/2)*sizeof(complex);
    complex* twiddle_factor_array = (complex*)malloc(twiddle_factor_array_size);
    precalcola_twiddle_factors(N, twiddle_factor_array);
    complex* d_twiddle_factor_array;
    hipMalloc(&d_twiddle_factor_array, twiddle_factor_array_size);
    hipMemcpy(d_twiddle_factor_array, twiddle_factor_array, twiddle_factor_array_size, hipMemcpyHostToDevice);

    // Configurazione dei blocchi e dei thread per gli stadi (in generale diversa da quella per il bit reversal)
    threads_per_block = 256;
    num_threads = N/2;  // per calcolare N campioni della trasformata, ho bisogno di soli N/2 thread data la simmetria
    num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
    

    // Lancia i kernel per ogni stadio
    for (int stadio = 1; stadio <= num_stadi; stadio++) {
        int N_stadio_corrente = 1 << stadio;
        int N_stadio_corrente_mezzi = N_stadio_corrente/2;
        int smem_total_size = N_stadio_corrente_mezzi*sizeof(complex);  // prova con 'N_stadio_corrente' e basta, così a cazzo
        int smem_per_block_size = smem_total_size/num_blocks;
        
        printf("\tsmem_total_size: %d bytes, smem_per_block_size: %d bytes\n", smem_total_size, smem_per_block_size);

        fft_stage<<<num_blocks, threads_per_block, smem_per_block_size>>>(d_output, N, N_stadio_corrente, N_stadio_corrente_mezzi, num_stadi, stadio, d_twiddle_factor_array);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Errore nel kernel fft_bit_reversal: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
    }

    hipMemcpy(output, d_output, N*sizeof(complex), hipMemcpyDeviceToHost);
    double elapsed_gpu = cpuSecond() - start;

    // cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_twiddle_factor_array);
    free(twiddle_factor_array);

    return elapsed_gpu;
}



int main(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: %s <file_name>\n", argv[0]);
        return 1;
    }

    const char* FILE_NAME = argv[1];
    drwav wav_in;
    
    if (!drwav_init_file(&wav_in, FILE_NAME, NULL)) {
        fprintf(stderr, "Errore nell'aprire il file %s.wav.\n", FILE_NAME);
        return 1;
    }

    size_t num_samples = wav_in.totalPCMFrameCount * wav_in.channels;
    printf("NUMERO DI CAMPIONI NEL FILE AUDIO SCELTO: %ld; -> %0.2f secondi\n\n", num_samples, (double)num_samples/SAMPLE_RATE);

    // importante avere una potenza di 2
    int padded_samples = 1 << (int)ceil(log2(num_samples));
    if (padded_samples > num_samples) {
        num_samples = padded_samples;
    }

    /*
        Alloco memoria per:
            - campioni PCM a 16 bit del file di ingresso
            - campioni PCM a 16 bit del file di ingresso convertiti in numeri complessi
            - campioni della trasformata ottenuti con FFT
    */
    short* signal_samples = (short*)malloc(num_samples * sizeof(short));
    if (signal_samples == NULL) {
        fprintf(stderr, "Errore nell'allocazione della memoria.\n");
        return 1;
    }
    complex* complex_signal_samples = (complex*)malloc(num_samples * sizeof(complex));
    if (complex_signal_samples == NULL) {
        fprintf(stderr, "Errore nell'allocazione della memoria.\n");
        return 1;
    }
    complex* fft_samples = (complex*)malloc(num_samples * sizeof(complex));
    if (fft_samples == NULL) {
        fprintf(stderr, "Errore nell'allocazione della memoria.\n");
        return 1;
    }

    // Lettura dei dati audio dal file di input
    size_t frames_read = drwav_read_pcm_frames_s16(&wav_in, wav_in.totalPCMFrameCount, signal_samples);
    if (frames_read != wav_in.totalPCMFrameCount) {
        fprintf(stderr, "Errore durante la lettura dei dati audio.\n");
        return 1;
    }
    drwav_uninit(&wav_in); 

    // calcolo la FFT
    convert_to_complex(signal_samples, complex_signal_samples, num_samples);
    double start = cpuSecond();
    fft_iterativa(complex_signal_samples, fft_samples, num_samples);
    double elapsed_host = cpuSecond() - start;









    /* ESECUZIONE CON GPU */











    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    
    complex* gpu_ref_fft_samples = (complex *)malloc(num_samples*sizeof(complex));
    
    double elapsed_device = fft_iterativa_cuda(complex_signal_samples, gpu_ref_fft_samples, num_samples); 
    
    checkResult(fft_samples, gpu_ref_fft_samples, num_samples);
    printf("Host: %f ms\n", elapsed_host*1000);
    printf("Device: %f ms\n", elapsed_device*1000);
    printf("SPEEDUP: %f\n", elapsed_host/elapsed_device);










    /* --- PARTE IFFT --- */

    

    // inizializzazione dati
    char generated_filename[100];   //dimensione arbitraria perchè non ho voglia
    sprintf(generated_filename, "GPU-IFFT-generated-%s", FILE_NAME);
    // mi assicuro di non imbrogliare ricopiando i dati di prima
    memset(signal_samples, 0, num_samples*sizeof(short));
    memset(complex_signal_samples, 0, num_samples);

    // Preparazione del formato del file di output
    drwav_data_format format_out;
    format_out.container = drwav_container_riff;
    format_out.format = DR_WAVE_FORMAT_PCM;
    format_out.channels = 1;              // Mono
    format_out.sampleRate = SAMPLE_RATE;  // Frequenza di campionamento
    format_out.bitsPerSample = 16;        // 16 bit per campione

    // Inizializzazione del file di output
    drwav wav_out;
    if (!drwav_init_file_write(&wav_out, generated_filename, &format_out, NULL)) {
        fprintf(stderr, "Errore nell'aprire il file di output %s.\n", generated_filename);
        return 1;
    }
    
    ifft_iterativa(gpu_ref_fft_samples, complex_signal_samples, num_samples);
    convert_to_short(complex_signal_samples, signal_samples, num_samples);

    // Scrittura dei dati audio nel file di output
    drwav_write_pcm_frames(&wav_out, num_samples, signal_samples);
    drwav_uninit(&wav_out); // Chiusura del file di output

    printf("File WAV %s creato con successo\n", generated_filename);

    free(signal_samples);
    free(complex_signal_samples);
    free(fft_samples);
    free(gpu_ref_fft_samples);
}