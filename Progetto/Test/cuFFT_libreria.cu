#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CHECK_CUFFT(call) \
{ \
    hipfftResult err = call; \
    if (err != HIPFFT_SUCCESS) { \
        printf("cuFFT error at %s:%d: %d\n", __FILE__, __LINE__, err); \
        exit(1); \
    } \
}

/*
    funzioni di utilità
*/

double cpuSecond() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return ((double)ts.tv_sec + (double)ts.tv_nsec * 1.e-9);
}


int main(int argc, char **argv) {
    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set-up dei campioni fittizzi
    const size_t N = 1 << 23; // Array size: 2^23
    const size_t bytes = N * sizeof(hipfftComplex);  // sono due float32

    hipfftComplex *h_data = (hipfftComplex*)malloc(bytes);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    for (size_t i = 0; i < N; ++i) {
        h_data[i].x = (float)(i % 256); // Real part
        h_data[i].y = 0.0f;            // Imaginary part
    }

    // Allocate device memory
    hipfftComplex *d_data;
    CHECK(hipMalloc((void**)&d_data, bytes));
    // Copy data to device
    CHECK(hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice));

    double start = cpuSecond();
    // Create cuFFT plan
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));
    // Execute FFT on the device
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    // Copy result back to host
    CHECK(hipMemcpy(h_data, d_data, bytes, hipMemcpyDeviceToHost));
    double elapsed = cpuSecond() - start;

    // Clean up
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK(hipFree(d_data));
    free(h_data);

    printf("tempo: %f ms\n", elapsed*1000);
}