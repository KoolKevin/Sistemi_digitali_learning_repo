#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

__global__ void reduceNeighbored(int *d_idata, int *d_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    // porzione di vettore che interessa ad un determinato blocco
    int *idata = d_idata + blockIdx.x * blockDim.x * 2;
    
    /* Unroll del loop sotto */
    // ogni thread aggiunge un elemento dal blocco dati vicino (tutti i thread del blocco alla prima riduzione sono attivi)
    if (idx + blockDim.x < n)
        d_idata[idx] += d_idata[idx + blockDim.x];

    __syncthreads();

     if (idx + blockDim.x/2 < n)
        d_idata[idx] += d_idata[idx + blockDim.x/2];

    __syncthreads();


    // Riduzione in-place nella memoria globale
    for (int stride = blockDim.x/4; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }

    if (tid == 0)
        d_odata[blockIdx.x] = idata[0]; // Il thread 0 scrive il risultato del blocco in g_odata
}


int main(int argc, char **argv) {
    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    
    // Dimensione dell’array (potenza di 2)
    int size = 1 << 26;
    // Configurazione Griglia e Blocchi
    int blocksize = 512;
    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    // Allocazione ed Inizializzazione Memoria Host
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x * sizeof(int));
    // Inizializzazione Random (max 10)
    for (int i = 0; i < size; i++)
        h_idata[i] = (int)(rand() % 10);

    // Allocazione Memoria Device
    int *d_idata, *d_odata;
    hipMalloc((void **)&d_idata, bytes);
    hipMalloc((void **)&d_odata, grid.x * sizeof(int));
    // Trasferimento Dati Host -> Device + Calcolo Parallelo
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);

    reduceNeighbored<<<grid.x/2, block>>>(d_idata, d_odata, size);
    // Sincronizzazione prima della Riduzione Globale
    hipDeviceSynchronize(); 

    // Trasferimento Risultati Device -> Host + Somma Finale
    int gpu_sum = 0;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];

    printf("GPU Reduction Sum: %d\n", gpu_sum);
}