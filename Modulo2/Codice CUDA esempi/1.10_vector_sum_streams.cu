#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define NUM_STREAMS 8

const int N = 1 << 20; 

__global__ void vectorAdd(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    hipStream_t streams[NUM_STREAMS];
    float *d_a[NUM_STREAMS], *d_b[NUM_STREAMS], *d_c[NUM_STREAMS];
    float *h_a, *h_b, *h_c[NUM_STREAMS]; 
    
    // Allocazione pinned memory per host
    hipHostMalloc(&h_a, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_b, N * sizeof(float), hipHostMallocDefault);
    for(int i = 0; i < NUM_STREAMS; i++) {
        hipHostMalloc(&h_c[i], N * sizeof(float), hipHostMallocDefault);
    }
    
    // Inizializzazione dati
    for(int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Creazione stream e allocazione memoria device
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
        hipMalloc(&d_a[i], N * sizeof(float));
        hipMalloc(&d_b[i], N * sizeof(float));
        hipMalloc(&d_c[i], N * sizeof(float));
        
        // Copia asincrona dei dati di input
        hipMemcpyAsync(d_a[i], h_a, N * sizeof(float), 
                       hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_b[i], h_b, N * sizeof(float), 
                       hipMemcpyHostToDevice, streams[i]);
        
        // Lancio del Kernel
        vectorAdd<<<1, 64, 0, streams[i]>>>(d_a[i], d_b[i], d_c[i], N);
        
        // Copia asincrona dei risultati
        hipMemcpyAsync(h_c[i], d_c[i], N * sizeof(float), 
                       hipMemcpyDeviceToHost, streams[i]);
    }
    
    // Sincronizzazione di tutti gli stream
    hipDeviceSynchronize();

    // Free
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipFree(d_a[i]);
        hipFree(d_b[i]);
        hipFree(d_c[i]);
        hipHostFree(h_c[i]);
        hipStreamDestroy(streams[i]);
    }
    hipHostFree(h_a);
    hipHostFree(h_b);

    return 0;
}
